#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include <algorithm>
#include <iterator>

#include "Colony.h"
#include <vector>
#include "Window.h"
#include <stdio.h>
//#include "SDL.h"
#undef main

#define WIDTH 1000
#define HEIGHT 700

// ---------------------------------------------------------

int id = 0;

__device__ int generate(hiprandState* globalState, int ind);
__global__ void setup_kernel(hiprandState* state, unsigned long seed);
__global__ void update(Ant* a, hiprandState* globalState, int w, int h);
hipError_t updateAnts(Colony *c);


int main()
{
    Window window("Ant Sim", WIDTH, HEIGHT);
    Colony c(Point(WIDTH / 2 + 1, HEIGHT / 2 + 1), 10, ++id);
    
    c.printInfo();
    c.printAnts();

    //updateAnts(&c);
    //std::cout << "\n------\n\n";
    //c.printAnts();

    for (int i = 0; i < 10; ++i) {
        updateAnts(&c);
        c.printAnts();
        std::cout << std::endl;
    }

    /*while (!window.isClosed()) {
        window.pollEvents();
        window.clear();
    }*/

    return 0;
}


__device__ int generate(hiprandState* globalState, int ind)
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    //float RANDOM = hiprand_uniform(&localState);
    int RANDOM = hiprand(&localState) % 3 - 1;
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel(hiprandState* state, unsigned long seed)
{
    int id = threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void update(Ant* a, hiprandState* globalState, int w, int h) {
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;   
    if (a[idx].getLifeSpan() > 0) {

        int n = generate(globalState, idx);
        //printf("%i\n", n);

        Point oldp = a[idx].getPos();

        a[idx].setPos(Point(oldp.getX() + n, oldp.getY() + n));
        a[idx].live();
    }
}


hipError_t updateAnts(Colony *c) {

    int size = c->getAntCount();

    Ant* dev_ants;
    hiprandState* devStates;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);

    // Allocate GPU buffers for the ants vector
    cudaStatus = hipMalloc((void**)&dev_ants, size * sizeof(Ant));
    cudaStatus = hipMalloc(&devStates, size * sizeof(hiprandState));    

    // Copy input vector from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_ants, c->ants, size * sizeof(Ant), hipMemcpyHostToDevice);    
    
    srand(time(0));
    int seed = rand();
    setup_kernel<<<1, size>>>(devStates,seed);

    update<<<1, size>>>(dev_ants, devStates, WIDTH, HEIGHT);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();    

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c->ants, dev_ants, size * sizeof(Ant), hipMemcpyDeviceToHost);
    

Error:
    hipFree(dev_ants);
    hipFree(devStates);

    return cudaStatus;
}


















/*

hipError_t updateAnts(Colony *c) {

    int size = c->getAntCount();

    Ant* dev_ants;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for the ants vector
    cudaStatus = hipMalloc((void**)&dev_ants, size * sizeof(Ant));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vector from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_ants, c->ants, size * sizeof(Ant), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy to, failed!");
        goto Error;
    }

    update<<<1, size>>>(dev_ants);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c->ants, dev_ants, size * sizeof(Ant), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy back, failed!");
        goto Error;
    }

Error:
    hipFree(dev_ants);

    return cudaStatus;
}

*/

/*



__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void update(uint3 *a) {
    int idx = threadIdx.x;

    a[idx].x = idx;
}

__global__ void update2(Point* a) {
    int idx = threadIdx.x;

    a[idx].setX(idx);
}

void runUpdate(uint3* a, int size) {

    uint3* dev_a;
    hipMalloc((void**)&dev_a, size * sizeof(uint3));

    hipMemcpy(dev_a, a, size * sizeof(uint3), hipMemcpyHostToDevice);

    update<<<1, size>>>(dev_a);

    hipMemcpy(a, dev_a, size * sizeof(uint3), hipMemcpyDeviceToHost);
}

void runUpdate2(Point* a, int size) {

    Point* dev_a;
    hipMalloc((void**)&dev_a, size * sizeof(Point));

    hipMemcpy(dev_a, a, size * sizeof(Point), hipMemcpyHostToDevice);

    update2<<<1, size >>> (dev_a);

    hipMemcpy(a, dev_a, size * sizeof(Point), hipMemcpyDeviceToHost);
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << < 1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

*/